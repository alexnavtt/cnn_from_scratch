#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include "cnn_from_scratch/Matrix/dim.h"
#include "cnn_from_scratch/CUDA/Matrix/convolve.cuh"

__global__ void kernel_test(my_cnn::Dim3 dim){
    printf("The value is (%d, %d, %d)\n", dim.x, dim.y, dim.z);
}

int main(){

    my_cnn::testConvolveReduce();

    std::cout << "=========\n";

    my_cnn::testConvolveDot();

    return 0;
}